#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <pthread.h>
#include <math.h>

#define ARRAY_SIZE 1000000
#define MAX_THREADS 16

typedef struct {
    double* arr1;
    double* arr2;
    double* result;
    int start;
    int end;
    int operation; // 1 - сложение, 2 - вычитание, 3 - умножение, 4 - деление
} ThreadData;

// Последовательные операции
void sequential_operations(double* arr1, double* arr2, double* result, int size, int operation) {
    for (int i = 0; i < size; i++) {
        switch (operation) {
            case 1: result[i] = arr1[i] + arr2[i]; break;
            case 2: result[i] = arr1[i] - arr2[i]; break;
            case 3: result[i] = arr1[i] * arr2[i]; break;
            case 4: result[i] = arr1[i] / arr2[i]; break;
        }
    }
}

// Функция потока для параллельных операций
void* parallel_operation(void* arg) {
    ThreadData* data = (ThreadData*)arg;
    for (int i = data->start; i < data->end; i++) {
        switch (data->operation) {
            case 1: data->result[i] = data->arr1[i] + data->arr2[i]; break;
            case 2: data->result[i] = data->arr1[i] - data->arr2[i]; break;
            case 3: data->result[i] = data->arr1[i] * data->arr2[i]; break;
            case 4: data->result[i] = data->arr1[i] / data->arr2[i]; break;
        }
    }
    return NULL;
}

// Параллельные операции
void parallel_operations(double* arr1, double* arr2, double* result, int size, 
                        int operation, int num_threads) {
    pthread_t threads[MAX_THREADS];
    ThreadData thread_data[MAX_THREADS];
    int chunk_size = size / num_threads;

    for (int i = 0; i < num_threads; i++) {
        thread_data[i].arr1 = arr1;
        thread_data[i].arr2 = arr2;
        thread_data[i].result = result;
        thread_data[i].start = i * chunk_size;
        thread_data[i].end = (i == num_threads - 1) ? size : (i + 1) * chunk_size;
        thread_data[i].operation = operation;

        pthread_create(&threads[i], NULL, parallel_operation, &thread_data[i]);
    }

    for (int i = 0; i < num_threads; i++) {
        pthread_join(threads[i], NULL);
    }
}

// Проверка результатов
int verify_results(double* seq_result, double* par_result, int size) {
    for (int i = 0; i < size; i++) {
        if (fabs(seq_result[i] - par_result[i]) > 1e-9) {
            printf("Ошибка проверки на индексе %d: %.6f != %.6f\n", 
                  i, seq_result[i], par_result[i]);
            return 0;
        }
    }
    return 1;
}

void test_operations(int size, int num_threads) {
    double* arr1 = malloc(size * sizeof(double));
    double* arr2 = malloc(size * sizeof(double));
    double* seq_result = malloc(size * sizeof(double));
    double* par_result = malloc(size * sizeof(double));

    // Инициализация массивов
    srand(time(NULL));
    for (int i = 0; i < size; i++) {
        arr1[i] = (double)rand() / RAND_MAX * 100.0 + 1.0; // От 1.0 до 101.0
        arr2[i] = (double)rand() / RAND_MAX * 100.0 + 1.0; // От 1.0 до 101.0
    }

    const char* operations[] = {"сложение", "вычитание", "умножение", "деление"};
    
    printf("\nТестирование для %d элементов (%d потоков):\n", size, num_threads);
    
    for (int op = 1; op <= 4; op++) {
        // Последовательная версия
        clock_t start = clock();
        sequential_operations(arr1, arr2, seq_result, size, op);
        double seq_time = (double)(clock() - start) / CLOCKS_PER_SEC;
        
        // Параллельная версия
        start = clock();
        parallel_operations(arr1, arr2, par_result, size, op, num_threads);
        double par_time = (double)(clock() - start) / CLOCKS_PER_SEC;
        
        // Проверка результатов
        int verified = verify_results(seq_result, par_result, size);
        
        printf("%10s: seq=%.6f сек, par=%.6f сек, ускорение=%.2fx, %s\n",
               operations[op-1], seq_time, par_time, seq_time/par_time,
               verified ? "верно" : "ошибка");
    }

    free(arr1);
    free(arr2);
    free(seq_result);
    free(par_result);
}

int main() {
    int sizes[] = {100000, 500000, 1000000};
    int num_tests = sizeof(sizes) / sizeof(sizes[0]);
    
    int thread_counts[] = {2, 4, 8, 16};
    int num_thread_counts = sizeof(thread_counts) / sizeof(thread_counts[0]);

    for (int i = 0; i < num_tests; i++) {
        for (int j = 0; j < num_thread_counts; j++) {
            test_operations(sizes[i], thread_counts[j]);
        }
    }

    return 0;
}
