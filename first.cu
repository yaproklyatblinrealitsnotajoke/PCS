#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <pthread.h>

#define ARRAY_SIZE 1000000
#define MAX_THREADS 16

// Структура для передачи данных в поток
typedef struct {
    int* array;
    int start;
    int end;
    long long partial_sum;
} ThreadData;

// Функция для вычисления части суммы в потоке
void* compute_partial_sum(void* arg) {
    ThreadData* data = (ThreadData*)arg;
    data->partial_sum = 0;
    
    for (int i = data->start; i < data->end; i++) {
        data->partial_sum += data->array[i];
    }
    
    return NULL;
}

// Последовательное вычисление суммы
long long sequential_sum(int* array, int size) {
    long long sum = 0;
    for (int i = 0; i < size; i++) {
        sum += array[i];
    }
    return sum;
}

// Параллельное вычисление суммы
long long parallel_sum(int* array, int size, int num_threads) {
    pthread_t threads[MAX_THREADS];
    ThreadData thread_data[MAX_THREADS];
    long long total_sum = 0;
    int chunk_size = size / num_threads;
    
    // Создание потоков
    for (int i = 0; i < num_threads; i++) {
        thread_data[i].array = array;
        thread_data[i].start = i * chunk_size;
        thread_data[i].end = (i == num_threads - 1) ? size : (i + 1) * chunk_size;
        
        pthread_create(&threads[i], NULL, compute_partial_sum, &thread_data[i]);
    }
    
    // Ожидание завершения потоков и суммирование результатов
    for (int i = 0; i < num_threads; i++) {
        pthread_join(threads[i], NULL);
        total_sum += thread_data[i].partial_sum;
    }
    
    return total_sum;
}

int main() {
    int* array = malloc(ARRAY_SIZE * sizeof(int));
    if (array == NULL) {
        fprintf(stderr, "Ошибка выделения памяти\n");
        return 1;
    }
    
    // Инициализация массива случайными числами
    srand(time(NULL));
    for (int i = 0; i < ARRAY_SIZE; i++) {
        array[i] = rand() % 100;
    }
    
    // Последовательное вычисление
    clock_t start = clock();
    long long seq_sum = sequential_sum(array, ARRAY_SIZE);
    clock_t end = clock();
    double seq_time = (double)(end - start) / CLOCKS_PER_SEC;
    
    printf("Последовательная сумма: %lld, время: %.6f сек.\n", seq_sum, seq_time);
    
    // Параллельное вычисление с разным количеством потоков
    int thread_counts[] = {2, 4, 8, 16};
    int num_tests = sizeof(thread_counts) / sizeof(thread_counts[0]);
    
    for (int i = 0; i < num_tests; i++) {
        int num_threads = thread_counts[i];
        
        start = clock();
        long long par_sum = parallel_sum(array, ARRAY_SIZE, num_threads);
        end = clock();
        double par_time = (double)(end - start) / CLOCKS_PER_SEC;
        
        printf("Параллельная сумма (%d потоков): %lld, время: %.6f сек., ускорение: %.2fx\n",
               num_threads, par_sum, par_time, seq_time / par_time);
    }
    
    free(array);
    return 0;
}
