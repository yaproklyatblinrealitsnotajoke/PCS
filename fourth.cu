
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <pthread.h>
#include <math.h>

#define ROWS 500
#define COLS 200
#define MAX_THREADS 16

typedef struct {
    double** arr1;
    double** arr2;
    double** result;
    int start_row;
    int end_row;
    int operation; // 1 - сложение, 2 - вычитание, 3 - умножение, 4 - деление
} ThreadData;

// Создание двумерного массива
double** create_2d_array(int rows, int cols) {
    double** arr = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++) {
        arr[i] = (double*)malloc(cols * sizeof(double));
    }
    return arr;
}

// Освобождение памяти двумерного массива
void free_2d_array(double** arr, int rows) {
    for (int i = 0; i < rows; i++) {
        free(arr[i]);
    }
    free(arr);
}

// Инициализация массива случайными значениями
void init_array(double** arr, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            arr[i][j] = (double)rand() / RAND_MAX * 100.0 + 1.0; // От 1.0 до 101.0
        }
    }
}

// Последовательные операции
void sequential_operations(double** arr1, double** arr2, double** result, 
                          int rows, int cols, int operation) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            switch (operation) {
                case 1: result[i][j] = arr1[i][j] + arr2[i][j]; break;
                case 2: result[i][j] = arr1[i][j] - arr2[i][j]; break;
                case 3: result[i][j] = arr1[i][j] * arr2[i][j]; break;
                case 4: result[i][j] = arr1[i][j] / arr2[i][j]; break;
            }
        }
    }
}

// Функция потока для параллельных операций
void* parallel_operation(void* arg) {
    ThreadData* data = (ThreadData*)arg;
    for (int i = data->start_row; i < data->end_row; i++) {
        for (int j = 0; j < COLS; j++) {
            switch (data->operation) {
                case 1: data->result[i][j] = data->arr1[i][j] + data->arr2[i][j]; break;
                case 2: data->result[i][j] = data->arr1[i][j] - data->arr2[i][j]; break;
                case 3: data->result[i][j] = data->arr1[i][j] * data->arr2[i][j]; break;
                case 4: data->result[i][j] = data->arr1[i][j] / data->arr2[i][j]; break;
            }
        }
    }
    return NULL;
}

// Параллельные операции
void parallel_operations(double** arr1, double** arr2, double** result, 
                        int rows, int cols, int operation, int num_threads) {
    pthread_t threads[MAX_THREADS];
    ThreadData thread_data[MAX_THREADS];
    int chunk_size = rows / num_threads;

    for (int i = 0; i < num_threads; i++) {
        thread_data[i].arr1 = arr1;
        thread_data[i].arr2 = arr2;
        thread_data[i].result = result;
        thread_data[i].start_row = i * chunk_size;
        thread_data[i].end_row = (i == num_threads - 1) ? rows : (i + 1) * chunk_size;
        thread_data[i].operation = operation;

        pthread_create(&threads[i], NULL, parallel_operation, &thread_data[i]);
    }

    for (int i = 0; i < num_threads; i++) {
        pthread_join(threads[i], NULL);
    }
}

// Проверка результатов
int verify_results(double** seq_result, double** par_result, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (fabs(seq_result[i][j] - par_result[i][j]) > 1e-9) {
                printf("Ошибка проверки на [%d][%d]: %.6f != %.6f\n", 
                      i, j, seq_result[i][j], par_result[i][j]);
                return 0;
            }
        }
    }
    return 1;
}

void test_operations(int rows, int cols, int num_threads) {
    double** arr1 = create_2d_array(rows, cols);
    double** arr2 = create_2d_array(rows, cols);
    double** seq_result = create_2d_array(rows, cols);
    double** par_result = create_2d_array(rows, cols);

    // Инициализация массивов
    srand(time(NULL));
    init_array(arr1, rows, cols);
    init_array(arr2, rows, cols);

    const char* operations[] = {"сложение", "вычитание", "умножение", "деление"};
    
    printf("\nТестирование для массива %dx%d (%d потоков):\n", rows, cols, num_threads);
    
    for (int op = 1; op <= 4; op++) {
        // Последовательная версия
        clock_t start = clock();
        sequential_operations(arr1, arr2, seq_result, rows, cols, op);
        double seq_time = (double)(clock() - start) / CLOCKS_PER_SEC;
        
        // Параллельная версия
        start = clock();
        parallel_operations(arr1, arr2, par_result, rows, cols, op, num_threads);
        double par_time = (double)(clock() - start) / CLOCKS_PER_SEC;
        
        // Проверка результатов
        int verified = verify_results(seq_result, par_result, rows, cols);
        
        printf("%10s: seq=%.6f сек, par=%.6f сек, ускорение=%.2fx, %s\n",
               operations[op-1], seq_time, par_time, seq_time/par_time,
               verified ? "верно" : "ошибка");
    }

    free_2d_array(arr1, rows);
    free_2d_array(arr2, rows);
    free_2d_array(seq_result, rows);
    free_2d_array(par_result, rows);
}

int main() {
    int sizes[][2] = {{500, 200}, {1000, 100}, {1000, 200}}; // Всего 100000, 100000, 200000 элементов
    int num_tests = sizeof(sizes) / sizeof(sizes[0]);
    
    int thread_counts[] = {2, 4, 8, 16};
    int num_thread_counts = sizeof(thread_counts) / sizeof(thread_counts[0]);

    for (int i = 0; i < num_tests; i++) {
        for (int j = 0; j < num_thread_counts; j++) {
            test_operations(sizes[i][0], sizes[i][1], thread_counts[j]);
        }
    }

    return 0;
}
