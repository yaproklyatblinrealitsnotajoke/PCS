#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <pthread.h>
#include <stdbool.h>

// Размеры массивов для тестирования
#define SIZE_1 100000
#define SIZE_2 500000
#define SIZE_3 1000000
#define MAX_THREADS 16

// Структура для передачи данных в поток Bitonic Sort
typedef struct {
    int* array;
    int start;
    int end;
    int dir; // 1 - по возрастанию, 0 - по убыванию
} BitonicData;

// Объединение двух отсортированных подмассивов (для Merge Sort)
void merge(int* array, int left, int mid, int right) {
    int n1 = mid - left + 1;
    int n2 = right - mid;

    int* L = malloc(n1 * sizeof(int));
    int* R = malloc(n2 * sizeof(int));

    for (int i = 0; i < n1; i++)
        L[i] = array[left + i];
    for (int j = 0; j < n2; j++)
        R[j] = array[mid + 1 + j];

    int i = 0, j = 0, k = left;
    while (i < n1 && j < n2) {
        if (L[i] <= R[j]) {
            array[k] = L[i];
            i++;
        } else {
            array[k] = R[j];
            j++;
        }
        k++;
    }

    while (i < n1) {
        array[k] = L[i];
        i++;
        k++;
    }

    while (j < n2) {
        array[k] = R[j];
        j++;
        k++;
    }

    free(L);
    free(R);
}

// Последовательная сортировка слиянием
void merge_sort(int* array, int left, int right) {
    if (left < right) {
        int mid = left + (right - left) / 2;
        merge_sort(array, left, mid);
        merge_sort(array, mid + 1, right);
        merge(array, left, mid, right);
    }
}

// Сравнение и обмен элементов для Bitonic Sort
void compare_and_swap(int* a, int* b, int dir) {
    if ((*a > *b && dir) || (*a < *b && !dir)) {
        int temp = *a;
        *a = *b;
        *b = temp;
    }
}

// Последовательная часть Bitonic Sort
void bitonic_merge(int* array, int low, int cnt, int dir) {
    if (cnt > 1) {
        int k = cnt / 2;
        for (int i = low; i < low + k; i++)
            compare_and_swap(&array[i], &array[i + k], dir);
        bitonic_merge(array, low, k, dir);
        bitonic_merge(array, low + k, k, dir);
    }
}

// Параллельная сортировка Bitonic Sort
void* bitonic_sort_thread(void* arg) {
    BitonicData* data = (BitonicData*)arg;
    if (data->end - data->start > 1) {
        int mid = (data->end + data->start) / 2;
        
        // Создаем потоки для каждой половины
        pthread_t thread1, thread2;
        
        BitonicData data1 = {data->array, data->start, mid, !data->dir};
        BitonicData data2 = {data->array, mid, data->end, data->dir};
        
        pthread_create(&thread1, NULL, bitonic_sort_thread, &data1);
        pthread_create(&thread2, NULL, bitonic_sort_thread, &data2);
        
        pthread_join(thread1, NULL);
        pthread_join(thread2, NULL);
        
        // Объединяем результаты
        bitonic_merge(data->array, data->start, data->end - data->start, data->dir);
    }
    return NULL;
}

// Проверка отсортированности массива
bool is_sorted(int* array, int size, int ascending) {
    for (int i = 0; i < size - 1; i++) {
        if (ascending && array[i] > array[i + 1])
            return false;
        if (!ascending && array[i] < array[i + 1])
            return false;
    }
    return true;
}

// Тестирование сортировки для массива заданного размера
void test_sort(int size) {
    int* array_seq = malloc(size * sizeof(int));
    int* array_par = malloc(size * sizeof(int));
    
    // Инициализация массива случайными числами
    srand(time(NULL));
    for (int i = 0; i < size; i++) {
        array_seq[i] = rand() % 10000;
        array_par[i] = array_seq[i];
    }
    
    printf("\nТестирование для массива из %d элементов:\n", size);
    
    // Последовательная сортировка слиянием
    clock_t start = clock();
    merge_sort(array_seq, 0, size - 1);
    clock_t end = clock();
    double seq_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Последовательная сортировка слиянием: %.6f сек., %s\n", 
           seq_time, is_sorted(array_seq, size, 1) ? "успешно" : "ошибка");
    
    // Параллельная Bitonic сортировка
    start = clock();
    BitonicData data = {array_par, 0, size, 1}; // Сортировка по возрастанию
    pthread_t main_thread;
    pthread_create(&main_thread, NULL, bitonic_sort_thread, &data);
    pthread_join(main_thread, NULL);
    end = clock();
    double par_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Параллельная Bitonic сортировка: %.6f сек., %s, ускорение: %.2fx\n", 
           par_time, is_sorted(array_par, size, 1) ? "успешно" : "ошибка",
           seq_time / par_time);
    
    free(array_seq);
    free(array_par);
}

int main() {
    // Тестируем для трех разных размеров массивов
    test_sort(SIZE_1);
    test_sort(SIZE_2);
    test_sort(SIZE_3);
    
    return 0;
}
